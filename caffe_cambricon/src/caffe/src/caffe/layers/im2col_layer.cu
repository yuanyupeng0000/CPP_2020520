/*
All modification made by Cambricon Corporation: © 2018-2019 Cambricon Corporation
All rights reserved.
All other contributions:
Copyright (c) 2014--2019, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/im2col_layer.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

template <typename Dtype>
void Im2colLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int num_kernels = channels_ * top[0]->count(channel_axis_ + 1);
  for (int n = 0; n < num_; ++n) {
    if (!force_nd_im2col_ && num_spatial_axes_ == 2) {
      im2col_gpu(bottom_data + n * bottom_dim_, channels_,
          bottom[0]->shape(channel_axis_ + 1),
          bottom[0]->shape(channel_axis_ + 2),
          kernel_shape_.cpu_data()[0], kernel_shape_.cpu_data()[1],
          pad_.cpu_data()[0], pad_.cpu_data()[1],
          stride_.cpu_data()[0], stride_.cpu_data()[1],
          dilation_.cpu_data()[0], dilation_.cpu_data()[1],
          top_data + n * top_dim_);
    } else {
      im2col_nd_gpu(bottom_data + n * bottom_dim_, num_spatial_axes_,
          num_kernels, bottom[0]->gpu_shape() + channel_axis_,
          top[0]->gpu_shape() + channel_axis_,
          kernel_shape_.gpu_data(), pad_.gpu_data(), stride_.gpu_data(),
          dilation_.gpu_data(), top_data + n * top_dim_);
    }
  }
}

template <typename Dtype>
void Im2colLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  for (int n = 0; n < num_; ++n) {
    if (!force_nd_im2col_ && num_spatial_axes_ == 2) {
      col2im_gpu(top_diff + n * top_dim_, channels_,
          bottom[0]->shape(channel_axis_ + 1),
          bottom[0]->shape(channel_axis_ + 2),
          kernel_shape_.cpu_data()[0], kernel_shape_.cpu_data()[1],
          pad_.cpu_data()[0], pad_.cpu_data()[1],
          stride_.cpu_data()[0], stride_.cpu_data()[1],
          dilation_.cpu_data()[0], dilation_.cpu_data()[1],
          bottom_diff + n * bottom_dim_);
    } else {
      col2im_nd_gpu(top_diff + n * top_dim_, num_spatial_axes_, bottom_dim_,
          bottom[0]->gpu_shape() + channel_axis_,
          top[0]->gpu_shape() + channel_axis_,
          kernel_shape_.gpu_data(), pad_.gpu_data(), stride_.gpu_data(),
          dilation_.gpu_data(), bottom_diff + n * bottom_dim_);
    }
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(Im2colLayer);

}  // namespace caffe
