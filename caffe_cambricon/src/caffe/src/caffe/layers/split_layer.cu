/*
All modification made by Cambricon Corporation: © 2018-2019 Cambricon Corporation
All rights reserved.
All other contributions:
Copyright (c) 2014--2019, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/split_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void SplitLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  for (int i = 0; i < top.size(); ++i) {
    top[i]->ShareData(*bottom[0]);
  }
}

template <typename Dtype>
void SplitLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) { return; }
  if (top.size() == 1) {
    caffe_copy(count_, top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff());
    return;
  }
  caffe_gpu_add(count_, top[0]->gpu_diff(), top[1]->gpu_diff(),
                bottom[0]->mutable_gpu_diff());
  // Add remaining top blob diffs.
  for (int i = 2; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_gpu_axpy(count_, Dtype(1.), top_diff, bottom_diff);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(SplitLayer);

}  // namespace caffe
