#include "hip/hip_runtime.h"
/*
All modification made by Cambricon Corporation: © 2018-2019 Cambricon Corporation
All rights reserved.
All other contributions:
Copyright (c) 2014--2019, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/concat_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Concat(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_concats, const int concat_size,
    const int top_concat_axis, const int bottom_concat_axis,
    const int offset_concat_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_concat_size = concat_size * bottom_concat_axis;
    const int concat_num = index / total_concat_size;
    const int concat_index = index % total_concat_size;
    const int top_index = concat_index +
        (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
    if (forward) {
      out_data[top_index] = in_data[index];
    } else {
      out_data[index] = in_data[top_index];
    }
  }
}

template <typename Dtype>
void ConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (bottom.size() == 1) { return; }
  Dtype* top_data = top[0]->mutable_gpu_data();
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = true;
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
    const int nthreads = bottom_concat_size * num_concats_;
    Concat<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_concats_, concat_input_size_,
        top_concat_axis, bottom_concat_axis, offset_concat_axis, top_data);
    offset_concat_axis += bottom_concat_axis;
  }
}

template <typename Dtype>
void ConcatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = false;
  for (int i = 0; i < bottom.size(); ++i) {
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    if (propagate_down[i]) {
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
      const int nthreads = bottom_concat_size * num_concats_;
      Concat<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
          nthreads, top_diff, kForward, num_concats_, concat_input_size_,
          top_concat_axis, bottom_concat_axis, offset_concat_axis, bottom_diff);
    }
    offset_concat_axis += bottom_concat_axis;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConcatLayer);

}  // namespace caffe
