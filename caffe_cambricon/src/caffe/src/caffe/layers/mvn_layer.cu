/*
All modification made by Cambricon Corporation: © 2018-2019 Cambricon Corporation
All rights reserved.
All other contributions:
Copyright (c) 2014--2019, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/mvn_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void MVNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int num;
  if (this->layer_param_.mvn_param().across_channels())
    num = bottom[0]->num();
  else
    num = bottom[0]->num() * bottom[0]->channels();

  int dim = bottom[0]->count() / num;

  // subtract mean
  caffe_gpu_gemv<Dtype>(CblasNoTrans, num, dim, 1. / dim, bottom_data,
      sum_multiplier_.gpu_data(), 0., mean_.mutable_gpu_data());  // EX
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, -1.,
      mean_.gpu_data(), sum_multiplier_.gpu_data(), 0.,
      temp_.mutable_gpu_data());
  caffe_gpu_add(temp_.count(), bottom_data, temp_.gpu_data(),
      top_data);  // X-EX

  if (this->layer_param_.mvn_param().normalize_variance()) {
    // compute variance using var(X) = E((X-EX)^2)
    caffe_gpu_powx(bottom[0]->count(), top_data, Dtype(2),
        temp_.mutable_gpu_data());  // (X-EX)^2
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num, dim, 1. / dim, temp_.gpu_data(),
        sum_multiplier_.gpu_data(), 0.,
        variance_.mutable_gpu_data());  // E((X-EX)^2)

    // normalize variance
    caffe_gpu_powx(variance_.count(), variance_.gpu_data(), Dtype(0.5),
          variance_.mutable_gpu_data());

    caffe_gpu_add_scalar(variance_.count(), eps_, variance_.mutable_gpu_data());

    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, 1.,
          variance_.gpu_data(), sum_multiplier_.gpu_data(), 0.,
          temp_.mutable_gpu_data());

    caffe_gpu_div(temp_.count(), top_data, temp_.gpu_data(), top_data);
  }
}

template <typename Dtype>
void MVNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  int num;
  if (this->layer_param_.mvn_param().across_channels())
    num = bottom[0]->num();
  else
    num = bottom[0]->num() * bottom[0]->channels();

  int dim = bottom[0]->count() / num;

  if (this->layer_param_.mvn_param().normalize_variance()) {
    caffe_gpu_mul(temp_.count(), top_data, top_diff, bottom_diff);
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num, dim, 1., bottom_diff,
          sum_multiplier_.gpu_data(), 0., mean_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, 1.,
          mean_.gpu_data(), sum_multiplier_.gpu_data(), 0.,
          bottom_diff);
    caffe_gpu_mul(temp_.count(), top_data, bottom_diff, bottom_diff);

    caffe_gpu_gemv<Dtype>(CblasNoTrans, num, dim, 1., top_diff,
            sum_multiplier_.gpu_data(), 0., mean_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, 1.,
            mean_.gpu_data(), sum_multiplier_.gpu_data(), 1.,
            bottom_diff);

    caffe_gpu_axpby(temp_.count(), Dtype(1), top_diff, Dtype(-1. / dim),
        bottom_diff);

    // put the squares of bottom into temp_
    caffe_gpu_powx(temp_.count(), bottom_data, Dtype(2),
        temp_.mutable_gpu_data());

    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, 1.,
        variance_.gpu_data(), sum_multiplier_.gpu_data(), 0.,
        temp_.mutable_gpu_data());

    caffe_gpu_div(temp_.count(), bottom_diff, temp_.gpu_data(), bottom_diff);
  } else {
    caffe_gpu_gemv<Dtype>(CblasNoTrans, num, dim, 1. / dim, top_diff,
            sum_multiplier_.gpu_data(), 0., mean_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, dim, 1, -1.,
            mean_.gpu_data(), sum_multiplier_.gpu_data(), 0.,
            temp_.mutable_gpu_data());
    caffe_gpu_add(temp_.count(), top_diff, temp_.gpu_data(), bottom_diff);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(MVNLayer);


}  // namespace caffe
