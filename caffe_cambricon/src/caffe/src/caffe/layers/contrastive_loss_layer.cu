#include "hip/hip_runtime.h"
/*
All modification made by Cambricon Corporation: © 2018-2019 Cambricon Corporation
All rights reserved.
All other contributions:
Copyright (c) 2014--2019, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <algorithm>
#include <vector>

#include "caffe/layers/contrastive_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void ContrastiveLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  Dtype margin = this->layer_param_.contrastive_loss_param().margin();
  bool legacy_version =
      this->layer_param_.contrastive_loss_param().legacy_version();
  Dtype loss(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
    if (static_cast<int>(bottom[2]->cpu_data()[i])) {  // similar pairs
      loss += dist_sq_.cpu_data()[i];
    } else {  // dissimilar pairs
      if (legacy_version) {
        loss += std::max(margin - dist_sq_.cpu_data()[i], Dtype(0.0));
      } else {
        Dtype dist = std::max(margin - sqrt(dist_sq_.cpu_data()[i]),
                              Dtype(0.0));
        loss += dist*dist;
      }
    }
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const bool legacy_version, const Dtype alpha,
    const Dtype* y, const Dtype* diff, const Dtype* dist_sq,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    if (static_cast<int>(y[n])) {  // similar pairs
      bottom_diff[i] = alpha * diff[i];
    } else {  // dissimilar pairs
      Dtype mdist(0.0);
      Dtype beta(0.0);
      if (legacy_version) {
        mdist = (margin - dist_sq[n]);
        beta = -alpha;
      } else {
        Dtype dist = sqrt(dist_sq[n]);
        mdist = (margin - dist);
        beta = -alpha * mdist / (dist + Dtype(1e-4)) * diff[i];
      }
      if (mdist > 0.0) {
        bottom_diff[i] = beta;
      } else {
        bottom_diff[i] = 0;
      }
    }
  }
}

template <typename Dtype>
void ContrastiveLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const int count = bottom[0]->count();
      const int channels = bottom[0]->channels();
      Dtype margin = this->layer_param_.contrastive_loss_param().margin();
      const bool legacy_version =
          this->layer_param_.contrastive_loss_param().legacy_version();
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, legacy_version, alpha,
          bottom[2]->gpu_data(),  // pair similarity 0 or 1
          diff_.gpu_data(),  // the cached eltwise difference between a and b
          dist_sq_.gpu_data(),  // the cached square distance between a and b
          bottom[i]->mutable_gpu_diff());
      CUDA_POST_KERNEL_CHECK;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ContrastiveLossLayer);

}  // namespace caffe
