#include "hip/hip_runtime.h"
/*
All modification made by Cambricon Corporation: © 2018-2019 Cambricon Corporation
All rights reserved.
All other contributions:
Copyright (c) 2014--2019, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/power_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void PowerLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // Special case where we can ignore the input: scale or power is 0.
  if (diff_scale_ == Dtype(0)) {
    Dtype value = (power_ == 0) ? Dtype(1) : pow(shift_, power_);
    caffe_gpu_set(count, value, top_data);
    return;
  }
  const Dtype* bottom_data = bottom[0]->gpu_data();
  caffe_copy(count, bottom_data, top_data);
  if (scale_ != Dtype(1)) {
    caffe_gpu_scal(count, scale_, top_data);
  }
  if (shift_ != Dtype(0)) {
    caffe_gpu_add_scalar(count, shift_, top_data);
  }
  if (power_ != Dtype(1)) {
    caffe_gpu_powx(count, top_data, power_, top_data);
  }
}

template <typename Dtype>
void PowerLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    if (diff_scale_ == Dtype(0) || power_ == Dtype(1)) {
      caffe_gpu_set(count, diff_scale_, bottom_diff);
    } else {
      const Dtype* bottom_data = bottom[0]->gpu_data();
      // Compute dy/dx = scale * power * (shift + scale * x)^(power - 1)
      //               = diff_scale * y / (shift + scale * x)
      if (power_ == Dtype(2)) {
        // Special case for y = (shift + scale * x)^2
        //     -> dy/dx = 2 * scale * (shift + scale * x)
        //              = diff_scale * shift + diff_scale * scale * x
        caffe_gpu_axpby(count, diff_scale_ * scale_, bottom_data,
            Dtype(0), bottom_diff);
        if (shift_ != Dtype(0)) {
          caffe_gpu_add_scalar(count, diff_scale_ * shift_, bottom_diff);
        }
      } else if (shift_ == Dtype(0)) {
        // Special case for y = (scale * x)^power
        //     -> dy/dx = scale * power * (scale * x)^(power - 1)
        //              = scale * power * (scale * x)^power * (scale * x)^(-1)
        //              = power * y / x
        const Dtype* top_data = top[0]->gpu_data();
        caffe_gpu_div(count, top_data, bottom_data, bottom_diff);
        caffe_gpu_scal(count, power_, bottom_diff);
      } else {
        caffe_copy(count, bottom_data, bottom_diff);
        if (scale_ != Dtype(1)) {
          caffe_gpu_scal(count, scale_, bottom_diff);
        }
        if (shift_ != Dtype(0)) {
          caffe_gpu_add_scalar(count, shift_, bottom_diff);
        }
        const Dtype* top_data = top[0]->gpu_data();
        caffe_gpu_div<Dtype>(count, top_data, bottom_diff, bottom_diff);
        if (diff_scale_ != Dtype(1)) {
          caffe_gpu_scal(count, diff_scale_, bottom_diff);
        }
      }
    }
    caffe_gpu_mul(count, top_diff, bottom_diff, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PowerLayer);


}  // namespace caffe
