/*
All modification made by Cambricon Corporation: © 2018-2019 Cambricon Corporation
All rights reserved.
All other contributions:
Copyright (c) 2014--2019, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/deconv_layer.hpp"

namespace caffe {

template <typename Dtype>
void DeconvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->backward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
  }
}

template <typename Dtype>
void DeconvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(top_diff + n * this->top_dim_,
              bottom_data + n * this->bottom_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->forward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_,
              this->param_propagate_down_[0]);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DeconvolutionLayer);

}  // namespace caffe
