#include "hip/hip_runtime.h"
/*
All modification made by Cambricon Corporation: © 2018-2019 Cambricon Corporation
All rights reserved.
All other contributions:
Copyright (c) 2014--2019, the respective contributors
All rights reserved.
For the list of contributors go to https://github.com/BVLC/caffe/blob/master/CONTRIBUTORS.md
Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:
    * Redistributions of source code must retain the above copyright notice,
      this list of conditions and the following disclaimer.
    * Redistributions in binary form must reproduce the above copyright
      notice, this list of conditions and the following disclaimer in the
      documentation and/or other materials provided with the distribution.
    * Neither the name of Intel Corporation nor the names of its contributors
      may be used to endorse or promote products derived from this software
      without specific prior written permission.
THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <vector>

#include "caffe/layers/slice_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Slice(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_slices, const int slice_size,
    const int bottom_slice_axis, const int top_slice_axis,
    const int offset_slice_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_slice_size = slice_size * top_slice_axis;
    const int slice_num = index / total_slice_size;
    const int slice_index = index % total_slice_size;
    const int bottom_index = slice_index +
        (slice_num * bottom_slice_axis + offset_slice_axis) * slice_size;
    if (forward) {
      out_data[index] = in_data[bottom_index];
    } else {
      out_data[bottom_index] = in_data[index];
    }
  }
}

template <typename Dtype>
void SliceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (top.size() == 1) { return; }
  int offset_slice_axis = 0;
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = true;
  for (int i = 0; i < top.size(); ++i) {
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, top_data);
    offset_slice_axis += top_slice_axis;
  }
}

template <typename Dtype>
void SliceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0] || top.size() == 1) { return; }
  int offset_slice_axis = 0;
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = false;
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, top_diff, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, bottom_diff);
    offset_slice_axis += top_slice_axis;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SliceLayer);

}  // namespace caffe
