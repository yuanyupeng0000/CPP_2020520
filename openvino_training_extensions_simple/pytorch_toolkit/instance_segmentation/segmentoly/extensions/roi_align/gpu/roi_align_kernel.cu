#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>


#define CUDA_1D_KERNEL_LOOP(i, n)                              \
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
         i += blockDim.x * gridDim.x)


/*** Forward ***/

__device__ float bilinear_interpolate(const float* bottom_data, const int height, const int width,
                                      float y, float x, const int index /* index for debug only*/) {
        // deal with cases that inverse elements are out of feature map boundary
        if (y < -1.0 || y > height || x < -1.0 || x > width) {
            // empty
            return 0;
        }

        if (y <= 0) {
            y = 0;
        }
        if (x <= 0) {
            x = 0;
        }

        int y_low = (int)y;
        int x_low = (int)x;
        int y_high;
        int x_high;

        if (y_low >= height - 1) {
            y_high = y_low = height - 1;
            y = (float)y_low;
        } else {
            y_high = y_low + 1;
        }

        if (x_low >= width - 1) {
            x_high = x_low = width - 1;
            x = (float)x_low;
        } else {
            x_high = x_low + 1;
        }

        float ly = y - y_low;
        float lx = x - x_low;
        float hy = 1. -ly, hx = 1. - lx;
        // do bilinear interpolation
        float v1 = bottom_data[y_low * width + x_low];
        float v2 = bottom_data[y_low * width + x_high];
        float v3 = bottom_data[y_high * width + x_low];
        float v4 = bottom_data[y_high * width + x_high];
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

        float val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

        return val;
    }

__global__ void ROIAlignForward(const int nthreads, const float* bottom_data, const float spatial_scale,
                                const int channels, const int height, const int width,
                                const int aligned_height, const int aligned_width, const int sampling_ratio,
                                const float* bottom_rois, float* top_data) {
    CUDA_1D_KERNEL_LOOP(index, nthreads) {
        // (n, c, ph, pw) is an element in the aligned output
        int pw = index % aligned_width;
        int ph = (index / aligned_width) % aligned_height;
        int c  = (index / aligned_width / aligned_height) % channels;
        int n  = index / aligned_width / aligned_height / channels;

        const float* offset_bottom_rois = bottom_rois + n * 5;
        int roi_batch_ind = offset_bottom_rois[0];

        // Do not using rounding; this implementation detail is critical
        float roi_start_w = offset_bottom_rois[1] * spatial_scale;
        float roi_start_h = offset_bottom_rois[2] * spatial_scale;
        float roi_end_w = offset_bottom_rois[3] * spatial_scale;
        float roi_end_h = offset_bottom_rois[4] * spatial_scale;

        // Force malformed ROIs to be 1x1
        float roi_width = fmaxf(roi_end_w - roi_start_w, 1.f);
        float roi_height = fmaxf(roi_end_h - roi_start_h, 1.f);
        float bin_size_h = roi_height / aligned_height;
        float bin_size_w = roi_width / aligned_width;

        const float* offset_bottom_data =
            bottom_data + (roi_batch_ind * channels + c) * height * width;

        // We use roi_bin_grid to sample the grid and mimic integral
        int roi_bin_grid_h = (sampling_ratio > 0)
            ? sampling_ratio
            : ceil(roi_height / aligned_height); // e.g., = 2
        int roi_bin_grid_w =
            (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / aligned_width);

        // We do average (integral) pooling inside a bin
        const float count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

        float output_val = 0.;
        for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
        {
            const float y = roi_start_h + ph * bin_size_h +
                (iy + .5f) * bin_size_h / roi_bin_grid_h;  // e.g., 0.5, 1.5
            for (int ix = 0; ix < roi_bin_grid_w; ix++) {
                const float x = roi_start_w + pw * bin_size_w +
                (ix + .5f) * bin_size_w / roi_bin_grid_w;

                float val = bilinear_interpolate(
                    offset_bottom_data, height, width, y, x, index);
                output_val += val;
            }
        }
        output_val /= count;

        top_data[index] = output_val;
    }
}


/*** Backward ***/
inline __device__ float gpu_atomic_add(const float val, float* address);
inline __device__ float gpu_atomic_add(const float val, float* address) {
  return atomicAdd(address, val);
}

__device__ void bilinear_interpolate_gradient(const int height, const int width, float y, float x,
                                              float& w1, float& w2, float& w3, float& w4,
                                              int& x_low, int& x_high, int& y_low, int& y_high,
                                              const int index /* index for debug only*/) {
    // deal with cases that inverse elements are out of feature map boundary
    if (y < -1.0 || y > height || x < -1.0 || x > width) {
        // empty
        w1 = w2 = w3 = w4 = 0.;
        x_low = x_high = y_low = y_high = -1;
        return;
    }

    if (y <= 0) {
        y = 0;
    }
    if (x <= 0) {
        x = 0;
    }

    y_low = (int)y;
    x_low = (int)x;

    if (y_low >= height - 1) {
        y_high = y_low = height - 1;
        y = (float)y_low;
    } else {
        y_high = y_low + 1;
    }

    if (x_low >= width - 1) {
        x_high = x_low = width - 1;
        x = (float)x_low;
    } else {
        x_high = x_low + 1;
    }

    float ly = y - y_low;
    float lx = x - x_low;
    float hy = 1. - ly, hx = 1. - lx;

    w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

    return;
}

__global__ void ROIAlignBackward(const int nthreads, const float* top_diff, const float spatial_scale,
                                 const int channels, const int height, const int width,
                                 const int aligned_height, const int aligned_width, const int sampling_ratio,
                                 float* bottom_diff, const float* bottom_rois) {
    CUDA_1D_KERNEL_LOOP(index, nthreads) {
        // (n, c, ph, pw) is an element in the aligned output
        int pw = index % aligned_width;
        int ph = (index / aligned_width) % aligned_height;
        int c  = (index / aligned_width / aligned_height) % channels;
        int n  = index / aligned_width / aligned_height / channels;

        const float* offset_bottom_rois = bottom_rois + n * 5;
        int roi_batch_ind = offset_bottom_rois[0];

        // Do not using rounding; this implementation detail is critical
        float roi_start_w = offset_bottom_rois[1] * spatial_scale;
        float roi_start_h = offset_bottom_rois[2] * spatial_scale;
        float roi_end_w = offset_bottom_rois[3] * spatial_scale;
        float roi_end_h = offset_bottom_rois[4] * spatial_scale;

        // Force malformed ROIs to be 1x1
        float roi_width = fmaxf(roi_end_w - roi_start_w, 1.f);
        float roi_height = fmaxf(roi_end_h - roi_start_h, 1.f);
        float bin_size_h = roi_height / aligned_height;
        float bin_size_w = roi_width / aligned_width;

        float* offset_bottom_diff =
            bottom_diff + (roi_batch_ind * channels + c) * height * width;

        int top_offset = (n * channels + c) * aligned_height * aligned_width;
        const float* offset_top_diff = top_diff + top_offset;
        const float top_diff_this_bin = offset_top_diff[ph * aligned_width + pw];

        // We use roi_bin_grid to sample the grid and mimic integral
        int roi_bin_grid_h = (sampling_ratio > 0)
            ? sampling_ratio
            : ceil(roi_height / aligned_height); // e.g., = 2
        int roi_bin_grid_w =
            (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / aligned_width);

        // We do average (integral) pooling inside a bin
        const float count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

        for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
        {
            const float y = roi_start_h + ph * bin_size_h +
                (iy + .5f) * bin_size_h / roi_bin_grid_h; // e.g., 0.5, 1.5
            for (int ix = 0; ix < roi_bin_grid_w; ix++) {
                const float x = roi_start_w + pw * bin_size_w +
                    (ix + .5f) * bin_size_w / roi_bin_grid_w;

                float w1, w2, w3, w4;
                int x_low, x_high, y_low, y_high;

                bilinear_interpolate_gradient(
                    height, width, y, x, w1, w2, w3, w4,
                    x_low, x_high, y_low, y_high, index);

                float g1 = top_diff_this_bin * w1 / count;
                float g2 = top_diff_this_bin * w2 / count;
                float g3 = top_diff_this_bin * w3 / count;
                float g4 = top_diff_this_bin * w4 / count;

                if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
                    gpu_atomic_add(g1, offset_bottom_diff + y_low * width + x_low);
                    gpu_atomic_add(g2, offset_bottom_diff + y_low * width + x_high);
                    gpu_atomic_add(g3, offset_bottom_diff + y_high * width + x_low);
                    gpu_atomic_add(g4, offset_bottom_diff + y_high * width + x_high);
                } // if
            } // ix
        } // iy
    } // CUDA_1D_KERNEL_LOOP
} // RoIAlignBackward

at::Tensor roi_align_forward_gpu(const at::Tensor& input,
                                 const at::Tensor& rois,
                                 const float spatial_scale,
                                 const int pooled_height,
                                 const int pooled_width,
                                 const int sampling_ratio) {
  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.type().is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = input.size(1);
  auto height = input.size(2);
  auto width = input.size(3);

  auto output = at::empty({num_rois, channels, pooled_height, pooled_width}, input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;

  dim3 grid(std::min(THCCeilDiv(output_size, 512L), 4096L));
  dim3 block(512);

  if (output.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return output;
  }

  ROIAlignForward<<<grid, block, 0>>>(
    output_size,
    input.contiguous().data<float>(),
    spatial_scale,
    channels,
    height,
    width,
    pooled_height,
    pooled_width,
    sampling_ratio,
    rois.contiguous().data<float>(),
    output.data<float>());

  THCudaCheck(hipGetLastError());
  return output;
}

at::Tensor roi_align_backward_gpu(const at::Tensor& grad,
                                  const at::Tensor& rois,
                                  const float spatial_scale,
                                  const int pooled_height,
                                  const int pooled_width,
                                  const int batch_size,
                                  const int channels,
                                  const int height,
                                  const int width,
                                  const int sampling_ratio) {
  AT_ASSERTM(grad.type().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.type().is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto grad_input = at::zeros({batch_size, channels, height, width}, grad.options());

  dim3 grid(std::min(THCCeilDiv(grad.numel(), 512L), 4096L));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return grad_input;
  }

  ROIAlignBackward<<<grid, block, 0>>>(
    grad.numel(),
    grad.contiguous().data<float>(),
    spatial_scale,
    channels,
    height,
    width,
    pooled_height,
    pooled_width,
    sampling_ratio,
    grad_input.data<float>(),
    rois.contiguous().data<float>());

  THCudaCheck(hipGetLastError());
  return grad_input;
}
